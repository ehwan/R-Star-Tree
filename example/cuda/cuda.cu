#include "hip/hip_runtime.h"
// must define for __host__ __device__ functions
#include <type_traits>
#define EH_RTREE_CUDA
#include <RTree.hpp>
#include <iostream>

#include <algorithm>
#include <memory>
#include <random>
#include <vector>

#include <hip/hip_runtime.h>

template <typename T>
struct cuda_managed_allocator
{
  using value_type = T;

  value_type* allocate(std::size_t n)
  {
    value_type* result;
    auto err = hipMallocManaged(&result, n * sizeof(value_type));
    if (err != hipSuccess)
    {
      std::cerr << hipGetErrorString(err) << std::endl;
      throw std::bad_alloc();
    }
    if (result == nullptr)
    {
      std::cerr << "Allocation failed\n";
      throw std::bad_alloc();
    }
    if ((intptr_t)(result) % alignof(value_type) != 0)
    {
      std::cerr << "Not aligned\n";
      throw std::bad_alloc();
    }
    std::cout << "Alloc " << n * sizeof(value_type) << " bytes\n";
    return result;
  }
  void deallocate(value_type* p, std::size_t n)
  {
    std::cout << "Dealloc " << n * sizeof(value_type) << " bytes\n";
    hipFree(p);
  }
};

namespace er = eh::rtree;
using rtree_type
    = er::RTree<er::aabb_t<float>, float, int, 4, 8, cuda_managed_allocator>;

__device__ __host__ void dfs(rtree_type::node_type* root, int leaf_level)
{
  if (leaf_level == 0)
  {
    printf("L: %d %lx\n", root->as_leaf()->size(), root);
    for (auto& c : *root->as_leaf())
    {
      printf("%d\n", c.second);
    }
  }
  else
  {
    printf("%d: %d %lx\n", leaf_level, root->as_node()->size(), root);
    for (auto& c : *root->as_node())
    {
      dfs(c.second->as_node(), leaf_level - 1);
    }
  }
}
__global__ void test_kernel(rtree_type::node_type* root, int leaf_level)
{
  dfs(root, leaf_level);
}

int main(int argc, char** argv)
{
  // 1-dimensional float RTree
  // geometry_traits< aabb_t<> > must be defined as __device__
  rtree_type managed_rtree;

  std::mt19937 generator { std::random_device {}() };
  std::uniform_real_distribution<float> dist01(0, 1);

  const int DATA_COUNT = 100;
  for (int i = 0; i < DATA_COUNT; ++i)
  {
    managed_rtree.insert({ dist01(generator), i });
  }

  dfs(managed_rtree.root(), managed_rtree.leaf_level());
  std::cout << "-------------------------\n";
  hipDeviceSynchronize();

  test_kernel<<<1, 1>>>(managed_rtree.root(), managed_rtree.leaf_level());
  hipDeviceSynchronize();

  return 0;
}