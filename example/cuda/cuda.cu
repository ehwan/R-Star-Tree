#include "hip/hip_runtime.h"
// must define for __host__ __device__ functions
#define EH_RTREE_CUDA
#include <RTree.hpp>

#include <algorithm>
#include <iostream>
#include <memory>
#include <random>
#include <type_traits>
#include <vector>

#include <hip/hip_runtime.h>

// Unified Memory Allocator
template <typename T>
struct cuda_managed_allocator
{
  using value_type = T;

  value_type* allocate(std::size_t n)
  {
    value_type* result;
    auto err = hipMallocManaged(&result, n * sizeof(value_type));
    if (err != hipSuccess)
    {
      std::cerr << hipGetErrorString(err) << std::endl;
      throw std::bad_alloc();
    }
    if (result == nullptr)
    {
      std::cerr << "Allocation failed\n";
      throw std::bad_alloc();
    }
    if ((intptr_t)(result) % alignof(value_type) != 0)
    {
      std::cerr << "Not aligned\n";
      throw std::bad_alloc();
    }
    return result;
  }
  void deallocate(value_type* p, std::size_t n)
  {
    hipFree(p);
  }
};

namespace er = eh::rtree;
using rtree_type
    = er::RTree<er::aabb_t<float>, float, int, 4, 8, cuda_managed_allocator>;

__device__ __host__ void dfs(rtree_type::node_type* root, int leaf_level)
{
  if (leaf_level == 0)
  {
    printf("Leaf: %d\n", root->as_leaf()->size());
    for (auto& c : *root->as_leaf())
    {
      printf("%d\n", c.second);
    }
  }
  else
  {
    printf("Level%d: %d\n", leaf_level, root->as_node()->size());
    for (auto& c : *root->as_node())
    {
      dfs(c.second->as_node(), leaf_level - 1);
    }
  }
}
__global__ void print_kernel(rtree_type::node_type* root, int leaf_level)
{
  dfs(root, leaf_level);
}

int main(int argc, char** argv)
{
  // 1-dimensional float RTree
  // geometry_traits< aabb_t<> > must be defined as __device__
  rtree_type managed_rtree;

  std::mt19937 generator { std::random_device {}() };
  std::uniform_real_distribution<float> dist01(0, 1);

  const int DATA_COUNT = 100;
  for (int i = 0; i < DATA_COUNT; ++i)
  {
    managed_rtree.insert({ dist01(generator), i });
  }

  dfs(managed_rtree.root(), managed_rtree.leaf_level());
  std::cout << "-------------------------\n";
  hipDeviceSynchronize();

  print_kernel<<<1, 1>>>(managed_rtree.root(), managed_rtree.leaf_level());
  hipDeviceSynchronize();

  return 0;
}